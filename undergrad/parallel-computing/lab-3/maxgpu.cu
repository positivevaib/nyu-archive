#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

unsigned int getmax(unsigned int *, unsigned int);

int main(int argc, char *argv[])
{
    unsigned int size = 0;  // The size of the array
    unsigned int i;  // loop index
    unsigned int * numbers; //pointer to the array
    
    if(argc !=2)
    {
        printf("usage: maxgpu num\n");
        printf("num = size of the array\n");
        exit(1);
    }
   
    size = atol(argv[1]);

    numbers = (unsigned int *)malloc(size * sizeof(unsigned int));
    if( !numbers )
    {
        printf("Unable to allocate mem for an array of size %u\n", size);
        exit(1);
    }    

    srand(time(NULL)); // setting a seed for the random number generator
    // Fill-up the array with random numbers from 0 to size-1 
    for( i = 0; i < size; i++)
        numbers[i] = rand()  % size;    
   
    printf(" The maximum number in the array is: %u\n", getmax(numbers, size));

    free(numbers);
    exit(0);
}

// kernel
__global__ void getmaxcu(unsigned int num[], unsigned int size, unsigned int offset) {
    __shared__ unsigned int block_num[1000];
    unsigned int t = threadIdx.x + (blockIdx.x * 1000);
    unsigned int boundary;

    if (offset != 1)
        offset = 1000;

    if (t < size) {
        block_num[threadIdx.x] = num[t*offset];

        boundary = 1000;
        if (t > (size - 1 - (size % 1000)))
            boundary = size % 1000;
        
        __syncthreads();

        while (boundary > 1) {
            if ((threadIdx.x < boundary/2) && (block_num[threadIdx.x] < block_num[threadIdx.x + (boundary+1)/2]))
                block_num[threadIdx.x] = block_num[threadIdx.x + (boundary+1)/2];

            boundary = (boundary+1)/2;

            __syncthreads();
        }

        if (threadIdx.x == 0)
            num[t] = block_num[0];
    }
}

/*
   input: pointer to an array of long int
          number of elements in the array
   output: the maximum number of the array
*/
unsigned int getmax(unsigned int num[], unsigned int size)
{
    unsigned int i;

    unsigned int * device_num;
    hipMalloc(&device_num, size * sizeof(unsigned int));
    hipMemcpy(device_num, num, size * sizeof(unsigned int), hipMemcpyHostToDevice);

    unsigned int threads_per_block = 1000;
    unsigned int tot_blocks = ceil((double)size/(threads_per_block));
    for (i = 0; i < ceil((double)log10(size)/log10(1000)); i++)
        getmaxcu<<<tot_blocks, threads_per_block>>>(device_num, (int)size*(1000/pow(1000, (i+1))), i+1);

    hipMemcpy(num, device_num, size * sizeof(unsigned int), hipMemcpyDeviceToHost);

    hipFree(device_num);

    return num[0];
}