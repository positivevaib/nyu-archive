#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <hip/hip_runtime.h>
#include <hipDNN.h>

#define C 3
#define H 1024
#define W 1024
#define P 1

#define K 64
#define FH 3
#define FW 3

#define NANO 1e9
#define PROGS 2
#define RUNS 5

#define CUDNN_CALL(x) do                                                                        \
{                                                                                               \
    hipdnnStatus_t ___s = (x);                                                                   \
    if (___s != HIPDNN_STATUS_SUCCESS)                                                           \
    {                                                                                           \
        fprintf(stderr, "%s:%d ERROR: %s\n", __FILE__,                                          \
                        __LINE__, hipdnnGetErrorString(___s));                                   \
        exit(-1);                                                                               \
    }                                                                                           \
} while (0);                                                    

// Forward kernel declaration
__global__ void convolution(double *, double *, double *, long, long, long, long, long);

int main(int argc, char * argv[])
{
    // Declare and initialize data structures
    // Initialize image (with and without padding) and filter arrays
    long PH = H + (2 * P);
    long PW = W + (2 * P);
    //-------------------------double hI[C][H][W], hPI[C][PH][PW], hF[K][C][FH][FW];
    double hI[C * H * W], hPI[C * PH * PW], hF[K * C * FH * FW];

    long c, x, y, k, i, j; 
    for (c = 0; c < C; c++)
    {
        for (x = 0; x < PH; x++)
            for (y = 0; y < PW; y++)
            {
                if (x == 0 || x == (PH - 1) || y == 0 || y == (PW - 1))
                    hPI[(c * PH * PW) + (x * PW) + y] = 0;//-----------------hPI[c][x][y] = 0;
                else
                {
                    hI[(c * H * W) + (x * W) + y] = c * (x + y);//----------------hI[c][x][y] = c * (x + y);
                    hPI[(c * PH * PW) + (x * PW) + y] = c * (x + y);//----------------hPI[c][x][y] = c * (x + y);
                }
            }

        for (k = 0; k < K; k++)
            for (i = 0; i < FH; i++)
                for (j = 0; j < FW; j++)
                    hF[(k * C * FH * FW) + (c * FH * FW) + (i * FW) + j] = (c + k) * (i + j);//-----------------------hF[k][c][i][j] = (c + k) * (i + j);
    }

    // Declare timespec structs and array to store time measurements
    struct timespec start, end;
    double ICopyTimes[PROGS * RUNS];
    double convTimes[PROGS * RUNS];
    double OCopyTimes[PROGS * RUNS];

    // Execute C1
    int prog = 0;
    for (i = 0; i < RUNS; i++)
    {
        // Declare host output array and device arrays
        double hO[K * H * W], * I, * F, * O;//--------------------------double hO[K][H][W], I[C][H + (2 * P)][W + (2 * P)], F[K][C][FH][FW], O[K][H][W];

        // Allocate device memory and transfer data to device
        hipMalloc(&I, sizeof(hPI));
        hipMalloc(&F, sizeof(hF));
        hipMalloc(&O, sizeof(hO));
        
        clock_gettime(CLOCK_MONOTONIC, &start);
        hipMemcpy(I, hPI, sizeof(hPI), hipMemcpyHostToDevice);
        clock_gettime(CLOCK_MONOTONIC, &end);

        ICopyTimes[(prog * RUNS) + i] = (end.tv_sec - start.tv_sec)                             \
                                        + ((end.tv_nsec - start.tv_nsec)/NANO);

        hipMemcpy(F, hF, sizeof(hF), hipMemcpyHostToDevice);

        // Set device grid and block dimensions
        dim3 dimGrid(K, H);
        dim3 dimBlock(W);

        // Invoke kernel
        clock_gettime(CLOCK_MONOTONIC, &start);
        convolution<<<dimGrid, dimBlock>>>(I, F, O, C, W, P, FH, FW);
        hipDeviceSynchronize();
        clock_gettime(CLOCK_MONOTONIC, &end);

        convTimes[(prog * RUNS) + i] = (end.tv_sec - start.tv_sec)                              \
                                       + ((end.tv_nsec - start.tv_nsec)/NANO);

        // Transfer output data from device
        clock_gettime(CLOCK_MONOTONIC, &start);
        hipMemcpy(hO, O, sizeof(hO), hipMemcpyDeviceToHost);
        clock_gettime(CLOCK_MONOTONIC, &end);

        OCopyTimes[(prog * RUNS) + i] = (end.tv_sec - start.tv_sec)                             \
                                        + ((end.tv_nsec - start.tv_nsec)/NANO);

        // Free device memory
        hipFree(I);
        hipFree(F);
        hipFree(O);

        // Compute checksums
        double ISum = 0;
        double OSum = 0;

        for (x = 0; x < H; x++)
            for (y = 0; y < W; y++)
            {
                for (c = 0; c < C; c++)
                    ISum += hPI[(c * PH * PW) + ((x + 1) * PW) + (y + 1)];//--------------------hPI[c][x + 1][y + 1];

                for (k = 0; k < K; k++)
                    OSum += hO[(k * H * W) + (x * W) + y];//----------------hO[k][x][y];
            }

        // Print output 
        printf("\nI = checksum: %lf                                                             \
                \nCopy host -> dev kernel: %lf s.                                               \
                \ntime kernel: %lf s.                                                           \
                \nCopy dev -> host kernel: %lf s.                                               \
                \nCUDA O = checksum: %lf",                                                      \
                ISum,                                                                           \
                ICopyTimes[(prog * RUNS) + i],                                                  \
                convTimes[(prog * RUNS) + i],                                                   \
                OCopyTimes[(prog * RUNS) + i],                                                  \
                OSum);
    }

    // Execute C2
    prog = 1;
    for (i = 0; i < RUNS; i++)
    {
        // Create cuDNN context
        hipdnnHandle_t cudnn;
        CUDNN_CALL(hipdnnCreate(&cudnn));

        // Create and configure descriptors
        // Input
        hipdnnTensorDescriptor_t inDesc;
        CUDNN_CALL(hipdnnCreateTensorDescriptor(&inDesc));
        CUDNN_CALL(hipdnnSetTensor4dDescriptor(inDesc,                                           \
                                              HIPDNN_TENSOR_NCHW,                                \
                                              HIPDNN_DATA_DOUBLE,                                \
                                              1, C, H, W));
    
        // Filter
        hipdnnFilterDescriptor_t filterDesc;
        CUDNN_CALL(hipdnnCreateFilterDescriptor(&filterDesc));
        CUDNN_CALL(hipdnnSetFilter4dDescriptor(filterDesc,                                       \
                                              HIPDNN_DATA_DOUBLE,                                \
                                              HIPDNN_TENSOR_NCHW,                                \
                                              K, C, FH, FW));
        
        // Output
        hipdnnTensorDescriptor_t outDesc;
        CUDNN_CALL(hipdnnCreateTensorDescriptor(&outDesc));
        CUDNN_CALL(hipdnnSetTensor4dDescriptor(outDesc,                                          \
                                              HIPDNN_TENSOR_NCHW,                                \
                                              HIPDNN_DATA_DOUBLE,                                \
                                              1, K, H, W));

        // Convolution
        hipdnnConvolutionDescriptor_t convDesc;
        CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&convDesc));
        CUDNN_CALL(hipdnnSetConvolution2dDescriptor(convDesc,                                    \
                                                   1, 1,                                        \
                                                   1, 1,                                        \
                                                   1, 1,                                        \
                                                   HIPDNN_CROSS_CORRELATION,                     \
                                                   HIPDNN_DATA_DOUBLE));                         \
        // Convolution algorithm
        hipdnnConvolutionFwdAlgo_t algo;
        CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(cudnn,                                   \
                                                      inDesc,                                   \
                                                      filterDesc,                               \
                                                      convDesc,                                 \
                                                      outDesc,                                  \
                                                      HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,     \
                                                      0,                                        \
                                                      &algo));

        // Determine device memory requirement
        size_t workspaceSize;
        CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,                               \
                                                           inDesc,                              \
                                                           filterDesc,                          \
                                                           convDesc,                            \
                                                           outDesc,                             \
                                                           algo,                                \
                                                           &workspaceSize));

        // Declare host output array, workspace and device arrays
        double hOut[K * H * W];//----------------double hOut[K][H][W];
        void * workspace, * in, * filter, * out;

        // Allocate device memory and transfer data to device
        hipMalloc(&workspace, workspaceSize);
        hipMalloc(&in, sizeof(hI));
        hipMalloc(&filter, sizeof(hF));
        hipMalloc(&out, sizeof(hOut));

        clock_gettime(CLOCK_MONOTONIC, &start);
        hipMemcpy(in, hI, sizeof(hI), hipMemcpyHostToDevice);
        clock_gettime(CLOCK_MONOTONIC, &end);
        
        ICopyTimes[(prog * RUNS) + i] = (end.tv_sec - start.tv_sec)                             \
                                        + ((end.tv_nsec - start.tv_nsec)/NANO);

        hipMemcpy(filter, hF, sizeof(hF), hipMemcpyHostToDevice);

        // Execute convolution
        const float alpha = 1;
        const float beta = 0;

        clock_gettime(CLOCK_MONOTONIC, &start);
        CUDNN_CALL(hipdnnConvolutionForward(cudnn,                                               \
                                           &alpha,                                              \
                                           inDesc, hI,                                          \
                                           filterDesc, hF,                                      \
                                           convDesc,                                            \
                                           algo,                                                \
                                           &workspace, workspaceSize,                           \
                                           &beta,                                               \
                                           outDesc, out));
        clock_gettime(CLOCK_MONOTONIC, &end);

        convTimes[(prog * RUNS) + i] = (end.tv_sec - start.tv_sec)                              \
                                       + ((end.tv_nsec - start.tv_nsec)/NANO);

        // Transfer data from device
        clock_gettime(CLOCK_MONOTONIC, &start);
        hipMemcpy(hOut, out, sizeof(hOut), hipMemcpyDeviceToHost);
        clock_gettime(CLOCK_MONOTONIC, &end);

        OCopyTimes[(prog * RUNS) + i] = (end.tv_sec - start.tv_sec)                             \
                                        + ((end.tv_nsec - start.tv_nsec)/NANO);

        // Free device memory and destroy descriptors and cuDNN context
        hipFree(workspace);
        hipFree(in);
        hipFree(filter);
        hipFree(out);

        CUDNN_CALL(hipdnnDestroyTensorDescriptor(inDesc));
        CUDNN_CALL(hipdnnDestroyFilterDescriptor(filterDesc));
        CUDNN_CALL(hipdnnDestroyTensorDescriptor(outDesc));
        CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(convDesc));

        CUDNN_CALL(hipdnnDestroy(cudnn));

        // Compute checksums
        double inSum = 0;
        double outSum = 0;

        for (x = 0; x < H; x++)
            for (y = 0; y < W; y++)
            {
                for (c = 0; c < C; c++)
                    inSum += hI[(c * H * W) + (x * W) + y];//--------------------hI[c][x][y];

                for (k = 0; k < K; k++)
                    outSum += hOut[(k * H * W) + (x * W) + y];//----------------------hOut[k][x][y];
            }

        // Print output 
        printf("\nI = checksum: %lf                                                             \
                \nCopy host -> dev kernel: %lf s.                                               \
                \ntime kernel: %lf s.                                                           \
                \nCopy dev -> host kernel: %lf s.                                               \
                \nCUDA O = checksum: %lf",                                                      \
                inSum,                                                                          \
                ICopyTimes[(prog * RUNS) + i],                                                  \
                convTimes[(prog * RUNS) + i],                                                   \
                OCopyTimes[(prog * RUNS) + i],                                                  \
                outSum);
    }

    // Compute averages
    double avg[PROGS] = {0, 0};
    for (i = 0; i < (PROGS * RUNS); i++)
        avg[i / RUNS] += (convTimes[i] / RUNS);

    // Print output
    printf("\n\n<Time>: Conv %lf s. cuDNN %lf s.\n", avg[0], avg[1]);

    return 0;
}

// Convolution kernel
__global__ void convolution(double * I, double * F, double * O, long C, long W, long P, long FH, long FW)
{
    // Compute image width with padding
    const long PH = H + (2 * P);
    const long PW = W + (2 * P);

    // Declare shared arrays
    __shared__ double ITile[C][FH][PW];
    __shared__ double FTile[C][FH][FW];

    // Transfer tiles to shared memory
    long c, h, w, i;
    for (c = 0; c < C; c++)
        for (h = 0; h < FH; h++)
        {
            i = threadIdx.x;
            while (i < PW)
            {
                ITile[c][h][i] = I[(c * PH * PW) + (((blockIdx.y * FH) + h) * PW) + i];//-------------------I[c][(blockIdx.y * FH) + h][i];

                if (i < FW)
                    FTile[c][h][i] = F[(blockIdx.x * C * FH * FW) + (c * FH * FW) + (h * FW) + i];//---------------------F[blockIdx.x][c][h][i];

                i += blockDim.x;
            }
        }
    __syncthreads();

    // Perform convolution
    double o = 0;
    for (c = 0; c < C; c++)
        for (h = 0; h < FH; h++)
            for (w = 0; w < FW; w++)
                o += ITile[c][h][(threadIdx.x * FW) + w] * FTile[c][h][w];

    O[(blockIdx.x * H * W) + (blockIdx.y * W) + threadIdx.x] = o;//---------------O[blockIdx.x][blockIdx.y][threadIdx.x] = o;
    __syncthreads();
}
