#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#include <hip/hip_runtime.h>
#include <hipDNN.h>


#define NANO 1e9

#define CUDNN_CALL(x) do                                                                        \
{                                                                                               \
    hipdnnStatus_t ___s = (x);                                                                   \
    if (___s != HIPDNN_STATUS_SUCCESS)                                                           \
    {                                                                                           \
        fprintf(stderr, "%s:%d ERROR: %s\n", __FILE__, __LINE__, hipdnnGetErrorString(___s));    \
        exit(-1);                                                                               \
    }                                                                                           \
} while(0);                                                                                     \


// Forward declarations
void init_I(double *, int, int, int);
void init_F(double *, int, int, int, int);

double get_checksum(double *, int, int, int);

double c1(int, double *, int, int, int, double *, int, int, int);
double c2(int, double *, int, int, int, double *, int, int, int); 

__device__ void convolve(int, double *, int, int, int, double *, int, int, double *);
__global__ void convolve_tiles_with_shared_mem(int, double *, int, int, int, double *, int, int, int, double *);


int main(int argc, char * argv[]) 
{
    // Initialize dimensions and I and F arrays
    int C = 3, H = 1024, W = 1024, P = 1;
    int K = 64, FH = 3, FW = 3;

    double I[C * H * W], F[K * C * FH * FW];
    init_I(I, C, H, W);
    init_F(F, K, C, FH, FW);

    // Execute programs and output results
    double c1_kernel_time = 0; 
    double c2_kernel_time = 0; 
    
    int runs = 5;
    
    int i;

    printf("C2");
    for (i = 0; i < runs; i++)
    {
        c2_kernel_time += c2(C, I, H, W, P, F, K, FH, FW);
    }

    printf("\n\nC1");
    for (i = 0; i < runs; i++)
    {
        c1_kernel_time += c1(C, I, H, W, P, F, K, FH, FW);
    }

    printf("\n\n<Time>: Conv %lf s. cuDNN %lf s.\n", c1_kernel_time / runs, c2_kernel_time / runs);
}


// Function to initialize I array
void init_I(double * I, int C, int H, int W)
{
    int c, h, w;
    for (c = 0; c < C; c++)
        for (h = 0; h < H; h++)
            for (w = 0; w < W; w++)
                I[(c * H * W) + (h * W) + w] = c * (h + w);
}


// Function to initialize F array
void init_F(double * F, int K, int C, int H, int W)
{
    int k, c, h, w;
    for (k = 0; k < K; k++)
        for (c = 0; c < C; c++)
            for (h = 0; h < H; h++)
                for (w = 0; w < W; w++)
                    F[(k * C * H * W) + (c * H * W) + (h * W) + w] = (c + k) * (h + w);
}


double c1(int C, double * I, int H, int W, int P, double * F, int K, int FH, int FW)
{
    // Determine array sizes, declare device arrays and allocate device memory
    size_t I_size = C * H * W * sizeof(double); 
    size_t F_size = K * C * FH * FW * sizeof(double); 
    size_t O_size = K * H * W * sizeof(double);

    double O[O_size], * dev_I, * dev_F, * dev_O;

    hipMalloc(&dev_I, I_size);
    hipMalloc(&dev_F, F_size);
    hipMalloc(&dev_O, O_size);

    struct timespec start, end;

    clock_gettime(CLOCK_MONOTONIC, &start);
    hipMemcpy(dev_I, I, I_size, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    clock_gettime(CLOCK_MONOTONIC, &end);

    double to_dev_time = (end.tv_sec - start.tv_sec) + ((end.tv_nsec - start.tv_nsec) / NANO);

    hipMemcpy(dev_F, F, F_size, hipMemcpyHostToDevice);
        
    // Set device properties and call kernel
    int block_size = 4;
    dim3 dimGrid(ceil(H / block_size), ceil(W / block_size));
    dim3 dimBlock(block_size, block_size, K);

    size_t tile_size = C * (block_size + (2 * P)) * (block_size + (2 * P));

    clock_gettime(CLOCK_MONOTONIC, &start);
    convolve_tiles_with_shared_mem<<<dimGrid, dimBlock, tile_size>>>(C, dev_I, H, W, P, dev_F, K, FH, FW, dev_O);
    hipDeviceSynchronize();
    clock_gettime(CLOCK_MONOTONIC, &end);

    double kernel_time = (end.tv_sec - start.tv_sec) + ((end.tv_nsec - start.tv_nsec) / NANO);

    // Copy output array to host, free device memory and output results
    clock_gettime(CLOCK_MONOTONIC, &start);
    hipMemcpy(O, dev_O, O_size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    clock_gettime(CLOCK_MONOTONIC, &end);

    double to_host_time = (end.tv_sec - start.tv_sec) + ((end.tv_nsec - start.tv_nsec) / NANO);

    hipFree(dev_I);
    hipFree(dev_F);
    hipFree(dev_O);

    double I_checksum = get_checksum(I, C, H, W);
    double O_checksum = get_checksum(O, K, H, W);

    printf("\n\nI = checksum: %lf\nCopy host -> dev kernel: %lf s.\ntime kernel: %lf s.\nCopy dev -> host kernel: %lf s.\nCUDA O = checksum:%lf", I_checksum, to_dev_time, kernel_time, to_host_time, O_checksum);

    return kernel_time;
}


double c2(int C, double * I, int H, int W, int P, double * F, int K, int FH, int FW)
{
    // Determine array sizes, declare device arrays and workspace and allocate device memory
    size_t I_size = C * H * W * sizeof(double); 
    size_t F_size = K * C * FH * FW * sizeof(double); 
    size_t O_size = K * H * W * sizeof(double);

    double O[O_size], * dev_I, * dev_F, * dev_O;
    void * workspace;

    hipMalloc(&dev_I, I_size);
    hipMalloc(&dev_F, F_size);
    hipMalloc(&dev_O, O_size);

    struct timespec start, end;

    clock_gettime(CLOCK_MONOTONIC, &start);
    hipMemcpy(dev_I, I, I_size, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    clock_gettime(CLOCK_MONOTONIC, &end);

    double to_dev_time = (end.tv_sec - start.tv_sec) + ((end.tv_nsec - start.tv_nsec) / NANO);

    hipMemcpy(dev_F, F, F_size, hipMemcpyHostToDevice);

    // Setup and execute CUDNN based convolution 
    hipdnnHandle_t cudnn;
    CUDNN_CALL(hipdnnCreate(&cudnn));

    hipdnnTensorDescriptor_t in_desc;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&in_desc));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(in_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, 1, C, H, W));

    hipdnnFilterDescriptor_t filter_desc;
    CUDNN_CALL(hipdnnCreateFilterDescriptor(&filter_desc));
    CUDNN_CALL(hipdnnSetFilter4dDescriptor(filter_desc, HIPDNN_DATA_DOUBLE, HIPDNN_TENSOR_NCHW, K, C, FH, FW));

    hipdnnTensorDescriptor_t out_desc;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, 1, K, H, W));

    hipdnnConvolutionDescriptor_t conv_desc;
    CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));
    CUDNN_CALL(hipdnnSetConvolution2dDescriptor(conv_desc, P, P, 1, 1, 1, 1, HIPDNN_CONVOLUTION, HIPDNN_DATA_DOUBLE));

    hipdnnConvolutionFwdAlgo_t conv_algo;
    CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(cudnn, in_desc, filter_desc, conv_desc, out_desc, HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &conv_algo));

    size_t workspace_size = 0;
    CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(cudnn, in_desc, filter_desc, conv_desc, out_desc, conv_algo, &workspace_size));
	hipMallocManaged(&workspace, workspace_size);

    double alpha = 1, beta = 0;

    clock_gettime(CLOCK_MONOTONIC, &start);
    CUDNN_CALL(hipdnnConvolutionForward(cudnn, &alpha, in_desc, dev_I, filter_desc, dev_F, conv_desc, conv_algo, workspace, workspace_size, &beta, out_desc, dev_O));
    hipDeviceSynchronize();
    clock_gettime(CLOCK_MONOTONIC, &end);

    double kernel_time = (end.tv_sec - start.tv_sec) + ((end.tv_nsec - start.tv_nsec) / NANO);

    // Copy output array to host, free device memory and output results
    clock_gettime(CLOCK_MONOTONIC, &start);
    hipMemcpy(O, dev_O, O_size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    clock_gettime(CLOCK_MONOTONIC, &end);

    double to_host_time = (end.tv_sec - start.tv_sec) + ((end.tv_nsec - start.tv_nsec) / NANO);

    hipFree(dev_I);
    hipFree(dev_F);
    hipFree(dev_O);
    hipFree(workspace);

	hipdnnDestroyTensorDescriptor(in_desc);
	hipdnnDestroyFilterDescriptor(filter_desc);
	hipdnnDestroyTensorDescriptor(out_desc);
	hipdnnDestroyConvolutionDescriptor(conv_desc);
	hipdnnDestroy(cudnn);

    double I_checksum = get_checksum(I, C, H, W);
    double O_checksum = get_checksum(O, K, H, W);

    printf("\n\nI = checksum: %lf\nCopy host -> dev kernel: %lf s.\ntime cudnn: %lf s.\nCopy dev -> host kernel: %lf s.\nCUDA O = checksum:%lf", I_checksum, to_dev_time, kernel_time, to_host_time, O_checksum);

    return kernel_time;
}


// Function to compute the sum of all elements of I and O arrays
double get_checksum(double * tensor, int C, int H, int W)
{
    double checksum = 0;

    int c, h, w;
    for (c = 0; c < C; c++)
        for (h = 0; h < H; h++)
            for (w = 0; w < W; w++)
                checksum += tensor[(c * H * W) + (h * W) + w];

    return checksum;
}


// CUDA kernel to perform convolution using tiles and shared memory
__global__ void convolve_tiles_with_shared_mem(int C, double * I, int H, int W, int P, double * F, int K, int FH, int FW, double * O)
{
    // Declare and populate tile array in shared memory 
    extern __shared__ double tile[];

    int TH = blockDim.x + (2 * P);
    int TW = blockDim.y + (2 * P);

    int k = threadIdx.z;
    int h = (blockIdx.x * blockDim.x) + threadIdx.x;
    int w = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (k == 0 && h < H && w < W) 
    {
        int th = threadIdx.x + P;
        int tw = threadIdx.y + P;

        int c;
        for (c = 0; c < C; c++)
        {
            tile[(c * TH * TW) + (th * TW) + tw] = I[(c * H * W) + (h * W) + w];

            int pad;

            for (pad = 1; pad <= P; pad++)
            {
                // Fill top rows and corners
                if (threadIdx.x == 0)
                {
                    if (h > 0)
                        tile[(c * TH * TW) + ((th - pad) * TW) + tw] = I[(c * H * W) + ((h - pad) * W) + w];
                    else
                        tile[(c * TH * TW) + ((th - pad) * TW) + tw] = 0;

                    // Top left corner
                    if (threadIdx.y == 0)
                    {
                        int h_pad, w_pad;
                        for (h_pad = pad; h_pad > 0; h_pad--)
                            for (w_pad = pad; w_pad > 0; w_pad--)
                            {
                                if (h > 0 && w > 0)
                                    tile[(c * TH * TW) + ((th - h_pad) * TW) + (tw - w_pad)] = I[(c * H * W) + ((h - h_pad) * W) + (w - w_pad)];
                                else
                                    tile[(c * TH * TW) + ((th - h_pad) * TW) + (tw - w_pad)] = 0;
                            }
                    }

                    // Top right corner
                    if (threadIdx.y == (blockDim.y - 1))
                    {
                        int h_pad, w_pad;
                        for (h_pad = pad; h_pad > 0; h_pad--)
                            for (w_pad = pad; w_pad > 0; w_pad--)
                            {
                                if (h > 0 && w < (W - 1))
                                    tile[(c * TH * TW) + ((th - h_pad) * TW) + (tw + w_pad)] = I[(c * H * W) + ((h - h_pad) * W) + (w + w_pad)];
                                else
                                    tile[(c * TH * TW) + ((th - h_pad) * TW) + (tw + w_pad)] = 0;
                            }
                    }
                }

                // Fill bottom rows and corners
                if (threadIdx.x == (blockDim.x - 1))
                {
                    if (h < (H - 1))
                        tile[(c * TH * TW) + ((th + pad) * TW) + tw] = I[(c * H * W) + ((h + pad) * W) + w];
                    else
                        tile[(c * TH * TW) + ((th + pad) * TW) + tw] = 0;

                    // Bottom left corner
                    if (threadIdx.y == 0)
                    {
                        int h_pad, w_pad;
                        for (h_pad = pad; h_pad > 0; h_pad--)
                            for (w_pad = pad; w_pad > 0; w_pad--)
                            {
                                if (h < (H - 1) && w > 0)
                                    tile[(c * TH * TW) + ((th + h_pad) * TW) + (tw - w_pad)] = I[(c * H * W) + ((h + h_pad) * W) + (w - w_pad)];
                                else
                                    tile[(c * TH * TW) + ((th + h_pad) * TW) + (tw - w_pad)] = 0;
                            }
                    }

                    // Bottom right corner
                    if (threadIdx.y == (blockDim.y - 1))
                    {
                        int h_pad, w_pad;
                        for (h_pad = pad; h_pad > 0; h_pad--)
                            for (w_pad = pad; w_pad > 0; w_pad--)
                            {
                                if (h < (H - 1) && w < (W - 1))
                                    tile[(c * TH * TW) + ((th + h_pad) * TW) + (tw + w_pad)] = I[(c * H * W) + ((h + h_pad) * W) + (w - w_pad)];
                                else
                                    tile[(c * TH * TW) + ((th + h_pad) * TW) + (tw + w_pad)] = 0;
                            }
                    }
                }

                // Fill left columns
                if (threadIdx.y == 0)
                {
                    if (w > 0)
                        tile[(c * TH * TW) + (th * TW) + (tw - pad)] = I[(c * H * W) + (h * W) + (w - pad)];
                    else
                        tile[(c * TH * TW) + (th * TW) + (tw - pad)] = 0;
                }

                // Fill right columns
                if (threadIdx.y == (blockDim.y - 1))
                {
                    if (w < (W - 1))
                        tile[(c * TH * TW) + (th * TW) + (tw + pad)] = I[(c * H * W) + (h * W) + (w + pad)];
                    else
                        tile[(c * TH * TW) + (th * TW) + (tw + pad)] = 0;
                }
            }
        }
    }
    __syncthreads();

    // Perform convolution
    convolve(C, tile, H, W, P, F, FH, FW, O);
}


// CUDA kernel to perform individual convolution computations
__device__ void convolve(int C, double * I, int H, int W, int P, double * F, int FH, int FW, double * O)
{
    double val = 0;

    int k = threadIdx.z; 
    int h = (blockIdx.x * blockDim.x) + threadIdx.x - P;
    int w = (blockIdx.y * blockDim.y) + threadIdx.y - P;

    int c, fh, fw, i, j;
    for (c = 0; c < C; c++)
        for (fh = 0; fh < FH; fh++)
        {
            i = h + fh;
            for (fw = 0; fw < FW; fw++)
            {
                j = w + fw;

                if (i < 0 || i >= H || j < 0 || j >= W)
                    continue;

                val += I[(c * H * W) + (i * W) + j] * F[(k * C * FH * FW) + (c * FH * FW) + ((FH - 1 - fh) * FW) + (FW - 1 - fw)];
            }
        }

    h += P;
    w += P;
    O[(k * H * W) + (h * W) + w] = val;
}

