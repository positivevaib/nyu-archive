#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <hip/hip_runtime.h>
#include <hipDNN.h>

#define NANO 1e9

#define CUDNN_CALL(x) do                                                                            \
{                                                                                                   \
    hipdnnStatus_t ___s = (x);                                                                       \
    if (___s != HIPDNN_STATUS_SUCCESS)                                                               \
    {                                                                                               \
        fprintf(stderr, "%s:%d ERROR: %s\n", __FILE__,                                              \
                        __LINE__, hipdnnGetErrorString(___s));                                       \
        exit(-1);                                                                                   \
    }                                                                                               \
} while (0);                                                    


// Forward declarations
typedef struct dimensions
{
    const int C, H, W, PH, PW;
    const int K, FH, FW;
};

void initArrs(double *, double *, double *, struct dimensions *);
void C1(struct dimensions *, double *, double *, double *, double *, double *);
void C2(struct dimensions *, double *, double *, double *, double *, double *);

__global__ void convKernel(double *, double *, double *,                                           \
                            int, int, int, int, int, int, int);


int main(int argc, char * argv[])
{
    // Initialize dimensions struct
    struct dimensions dims = {3, 1024, 1024, 1026, 1026, 64, 3, 3};
    
    // Initialize image (with and without padding) and filter arrays
    double hostI[dims.C * dims.H * dims.W], hostPI[dims.C * dims.PH * dims.PW], hostF[dims.K * dims.C * dims.FH * dims.FW];
    initArrs(hostI, hostPI, hostF, &dims);

    // Declare arrays to store time measurements
    double copyToDevTimes[10], convTimes[10], copyToHostTimes[10];

    // Execute programs
    printf("C1");
    C1(&dims, hostPI, hostF, copyToDevTimes, convTimes, copyToHostTimes);

    printf("\n\nC2");
    C2(&dims, hostI, hostF, copyToDevTimes, convTimes, copyToHostTimes);

    // Compute averages
    double avg[2];

    int i;
    for (i = 0; i < 2; i++)
        avg[i] = 0;

    for (i = 0; i < (2 * 5); i++)
        avg[i / 5] += (convTimes[i] / 5);

    // Print output
    printf("\n\n<Time>: Conv %lf s. cuDNN %lf s.\n", avg[0], avg[1]);

    return 0;
}


void initArrs(double * hostI, double * hostPI, double * hostF, struct dimensions * dims)
{
    int c, x, y, k, i, j; 
    for (c = 0; c < dims->C; c++)
    {
        for (x = 0; x < dims->PH; x++)
            for (y = 0; y < dims->PH; y++)
                hostPI[(c * dims->PH * dims->PW) + (x * dims->PW) + y] = 0;

        for (x = 0; x < dims->H; x++)
            for (y = 0; y < dims->W; y++)
            {
                hostI[(c * dims->H * dims->W) + (x * dims->W) + y] = c * (x + y);
                hostPI[(c * dims->PH * dims->PW) + ((x + 1) * dims->PW) + (y + 1)] = c * (x + y);
            }

        for (k = 0; k < dims->K; k++)
            for (i = 0; i < dims->FH; i++)
                for (j = 0; j < dims->FW; j++)
                    hostF[(k * dims->C * dims->FH * dims->FW) + (c * dims->FH * dims->FW) + (j * dims->FW) + i] = (c + k) * (i + j);
    }
}


void C1(struct dimensions * dims, double * hostPI, double * hostF, double * copyToDevTimes, double * convTimes, double * copyToHostTimes)
{
    // Declare timespec structs 
    struct timespec start, end;

    // Declare host output array and device arrays
    double hostO[dims->K * dims->H * dims->W], * devPI, * devF, * devO;

    int i, x, y, c, k;
    for (i = 0; i < 5; i++)
    {
        // Allocate device memory and transfer data to device
        hipMalloc(&devPI, sizeof(hostPI));
        hipMalloc(&devF, sizeof(hostF));
        hipMalloc(&devO, sizeof(hostO));
        
        clock_gettime(CLOCK_MONOTONIC, &start);
        hipMemcpy(devPI, hostPI, sizeof(hostPI), hipMemcpyHostToDevice);
        clock_gettime(CLOCK_MONOTONIC, &end);

        copyToDevTimes[i] = (end.tv_sec - start.tv_sec) + ((end.tv_nsec - start.tv_nsec)/NANO);

        hipMemcpy(devF, hostF, sizeof(hostF), hipMemcpyHostToDevice);

        // Set device grid and block dimensions
        dim3 dimGrid(dims->K, dims->H);
        dim3 dimBlock(dims->W / 2);

        // Invoke kernel
        clock_gettime(CLOCK_MONOTONIC, &start);
        convKernel<<<dimGrid, dimBlock, ((dims->C * dims->FH * (dims->PW / 2)) + (dims->C * dims->FH * dims->FW)) * sizeof(double)>>>(devPI, devF, devO, dims->C, dims->H, dims->W, dims->PH, dims->PW, dims->FH, dims->FW);
        hipDeviceSynchronize();
        clock_gettime(CLOCK_MONOTONIC, &end);

        convTimes[i] = (end.tv_sec - start.tv_sec) + ((end.tv_nsec - start.tv_nsec)/NANO);

        // Transfer output data from device
        clock_gettime(CLOCK_MONOTONIC, &start);
        hipMemcpy(hostO, devO, sizeof(hostO), hipMemcpyDeviceToHost);
        clock_gettime(CLOCK_MONOTONIC, &end);

        copyToHostTimes[i] = (end.tv_sec - start.tv_sec) + ((end.tv_nsec - start.tv_nsec)/NANO);

        // Free device memory
        hipFree(devPI);
        hipFree(devF);
        hipFree(devO);

        // Compute checksums
        double checksumPI = 0;
        for (c = 0; c < dims->C; c++)
            for (x = 0; x < dims->PH; x++)
                for (y = 0; y < dims->PW; y++)
                    checksumPI += hostPI[(c * dims->PH * dims->PW) + (x * dims->PW) + y];

        double checksumO = 0;
        for (k = 0; k < dims->K; k++)
            for (x = 0; x < dims->H; x++)
                for (y = 0; y < dims->W; y++)
                    checksumO += hostO[(k * dims->H * dims->W) + (x * dims->W) + y];

        // Print output 
        printf("\n\nI = checksum: %lf\nCopy host -> dev kernel: %lf s.\ntime kernel: %lf s.\nCopy dev -> host kernel: %lf s.\nCUDA O = checksum: %lf", checksumPI, copyToDevTimes[i], convTimes[i], copyToHostTimes[i], checksumO);
    }
}

void C2(struct dimensions * dims, double * hostI, double * hostF, double * copyToDevTimes, double * convTimes, double * copyToHostTimes)
{
    // Declare timespec structs 
    struct timespec start, end;

    // Declare host output array, workspace and device arrays
    double hostO[dims->K * dims->H * dims->W];
    void * workspace, * devI, * devF, * devO;

    int i, x, y, c, k;
    for (i = 0; i < 5; i++)
    {
        // Create cuDNN context
        hipdnnHandle_t cudnn;
        CUDNN_CALL(hipdnnCreate(&cudnn));

        // Create and configure descriptors
        // Input
        hipdnnTensorDescriptor_t descI;
        CUDNN_CALL(hipdnnCreateTensorDescriptor(&descI));
        CUDNN_CALL(hipdnnSetTensor4dDescriptor(descI, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, 1, dims->C, dims->H, dims->W));
    
        // Filter
        hipdnnFilterDescriptor_t descF;
        CUDNN_CALL(hipdnnCreateFilterDescriptor(&descF));
        CUDNN_CALL(hipdnnSetFilter4dDescriptor(descF, HIPDNN_DATA_DOUBLE, HIPDNN_TENSOR_NCHW, dims->K, dims->C, dims->FH, dims->FW));
        
        // Output
        hipdnnTensorDescriptor_t descO;
        CUDNN_CALL(hipdnnCreateTensorDescriptor(&descO));
        CUDNN_CALL(hipdnnSetTensor4dDescriptor(descO, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, 1, dims->K, dims->H, dims->W));

        // Convolution
        hipdnnConvolutionDescriptor_t descConv;
        CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&descConv));
        CUDNN_CALL(hipdnnSetConvolution2dDescriptor(descConv, 1, 1, 1, 1, 1, 1, HIPDNN_CONVOLUTION, HIPDNN_DATA_DOUBLE));                             

        // Convolution algorithm
        hipdnnConvolutionFwdAlgo_t algo;
        CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(cudnn, descI, descF, descConv, descO, HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algo));

        // Determine device memory requirement
        size_t workspaceSize;
        CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(cudnn, descI, descF, descConv, descO, algo, &workspaceSize));

        // Allocate device memory and transfer data to device
        hipMalloc(&workspace, workspaceSize);
        hipMalloc(&devI, sizeof(hostI));
        hipMalloc(&devF, sizeof(hostF));
        hipMalloc(&devO, sizeof(hostO));

        clock_gettime(CLOCK_MONOTONIC, &start);
        hipMemcpy(devI, hostI, sizeof(hostI), hipMemcpyHostToDevice);
        clock_gettime(CLOCK_MONOTONIC, &end);
        
        copyToDevTimes[5 + i] = (end.tv_sec - start.tv_sec) + ((end.tv_nsec - start.tv_nsec)/NANO);

        hipMemcpy(devF, hostF, sizeof(hostF), hipMemcpyHostToDevice);

        // Execute convolution
        const float alpha = 1;
        const float beta = 0;

        clock_gettime(CLOCK_MONOTONIC, &start);
        CUDNN_CALL(hipdnnConvolutionForward(cudnn, &alpha, descI, devI, descF, devF, descConv, algo, &workspace, workspaceSize, &beta, descO, devO));
        clock_gettime(CLOCK_MONOTONIC, &end);

        convTimes[5 + i] = (end.tv_sec - start.tv_sec) + ((end.tv_nsec - start.tv_nsec)/NANO);

        // Transfer data from device
        clock_gettime(CLOCK_MONOTONIC, &start);
        hipMemcpy(hostO, devO, sizeof(hostO), hipMemcpyDeviceToHost);
        clock_gettime(CLOCK_MONOTONIC, &end);

        copyToHostTimes[5 + i] = (end.tv_sec - start.tv_sec) + ((end.tv_nsec - start.tv_nsec)/NANO);

        // Free device memory and destroy descriptors
        hipFree(workspace);
        hipFree(devI);
        hipFree(devF);
        hipFree(devO);

        hipdnnDestroyTensorDescriptor(descI);
        hipdnnDestroyFilterDescriptor(descF);
        hipdnnDestroyTensorDescriptor(descO);
        hipdnnDestroyConvolutionDescriptor(descConv);

        hipdnnDestroy(cudnn);

        // Compute checksums
        double checksumI = 0;
        for (c = 0; c < dims->C; c++)
            for (x = 0; x < dims->H; x++)
                for (y = 0; y < dims->W; y++)
                    checksumI += hostI[(c * dims->H * dims->W) + (x * dims->W) + y];

        double checksumO = 0;
        for (k = 0; k < dims->K; k++)
            for (x = 0; x < dims->H; x++)
                for (y = 0; y < dims->W; y++)
                    checksumO += hostO[(k * dims->H * dims->W) + (x * dims->W) + y];

        // Print output 
        printf("\n\nI = checksum: %lf\nCopy host -> dev kernel: %lf s.\ntime kernel: %lf s.\nCopy dev -> host kernel: %lf s.\nCUDA O = checksum: %lf", checksumI, copyToDevTimes[i], convTimes[i], copyToHostTimes[i], checksumO);
    }
}

// Convolution kernel
__global__ void convKernel(double * PI, double * F, double * O, int C, int H, int W, int PH, int PW, int FH, int FW)
{
    // Declare shared arrays
    extern __shared__ double tile[];
    double * tilePI = tile;
    double * tileF = (double *) &tilePI[C * FH * (PW / 2)];

    // Transfer tiles to shared memory
    int half, c, h, w, i;
    for (half = 0; half < 2; half++)
    {
        for (c = 0; c < C; c++)
            for (h = 0; h < FH; h++)
            {
                i = threadIdx.x;
                while (i < (PW / 2))
                {
                    tilePI[(c * FH * (PW / 2)) + (h * (PW / 2)) + i] = PI[(c * PH * PW) + (((blockIdx.y * FH) + h) * PW) + ((half * blockDim.x) + i)];

                    if (i < FW)
                        tileF[(c * FH * FW) + (h * FW) + i] = F[(blockIdx.x * C * FH * FW) + (c * FH * FW) + (h * FW) + i];

                    i += blockDim.x;
                }
            }
        __syncthreads();

        // Perform convolution
        double o = 0;

        for (c = 0; c < C; c++)
            for (h = 0; h < FH; h++)
                for (w = 0; w < FW; w++)
                    o += tilePI[(c * FH * (PW / 2)) + (h * (PW / 2)) + ((threadIdx.x * FW) + w)] * tileF[(c * FH * FW) + (h * FW) + w];

        O[(blockIdx.x * H * W) + (blockIdx.y * W) + ((half * blockDim.x) + threadIdx.x)] = o;
        __syncthreads();
    }
}
